#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

using namespace std;

#define N 4
#define BLOCK_SIZE 32

__host__ void initProc(int *a, int *b, int len);
__global__ void kernelProc(int *a, int *b, int len);

int main(void) {
    int a[N + 2], b[N];

    cout << "A ";
    for(int i = 0; i < N + 2; i++) {
        a[i] = rand() % 100;
        cout << a[i] << " ";
    }

    initProc(a, b, N);

    cout << "\nResult: ";
    for(int i = 0; i < N; i++)
        cout << b[i] << " ";

    return 0;
}

__host__ void initProc(int *a, int *b, int len) {
    int* devA, *devB;

    size_t sizeA = (len + 2) * sizeof(int);
    size_t sizeB = len * sizeof(int);

    hipMalloc(&devA, sizeA);
    hipMalloc(&devB, sizeB);

    hipMemcpy(devA, a, sizeA, hipMemcpyHostToDevice);

    kernelProc<<<BLOCK_SIZE, BLOCK_SIZE>>>(devA, devB, len);

    hipMemcpy(b, devB, sizeB, hipMemcpyDeviceToHost);

    hipFree(devA);
    hipFree(devB);
}

__global__ void kernelProc(int *a, int *b, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid > len) return;

    __shared__ int sh[BLOCK_SIZE];
    sh[threadIdx.x] = a[tid + 1];

    __syncthreads();

    int pom;
    if(threadIdx.x == 0)
       pom = 3 * a[tid] + 10 * sh[threadIdx.x] + 7 * sh[threadIdx.x + 1];
    else if(threadIdx.x == blockDim.x - 1)
        pom = 3 * sh[threadIdx.x - 1] + 10 * sh[threadIdx.x] + 7 * a[tid + 2];
    else
        pom = 3 * sh[threadIdx.x - 1] + 10 * sh[threadIdx.x] + 7 * sh[threadIdx.x + 1];

    b[tid] = pom / 20.f;
}