#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<stdlib.h>

using namespace std;

#define N 100

/*
Da li je validno rešenje? -jeste.
Da li je najoptimalnije? -nije.
*/

__host__ void init(float* a, float * vec, int c);
__device__ float operatorX(int a, int b, float c);
__global__ void kernel(float* dev_a, float* out_vec, int* c);

int main(void) {
    float a[N][N];
    float vec[N];
    int c = 5;

    init(&a[0][0], &vec[0], c);

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            cout << a[i][j] << " ";
        }
        cout << endl;
    }
    cout << endl;

    for(int i = 0; i < N; i++) {
        cout << vec[i] << " ";
    }
    cout << endl;

    return 0;
}

__host__ void init(float* a, float * vec, int c) {
    float *dev_a, *dev_vec;
    int *dev_c;

    size_t a_size = N * N * sizeof(float);
    size_t vec_size = N * sizeof(float);

    hipMalloc(&dev_a, a_size);
    hipMalloc(&dev_vec, vec_size);
    hipMalloc(&dev_c, sizeof(int));

    hipMemcpy(dev_a, a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<N, 256>>>(dev_a, dev_vec, dev_c);

    hipMemcpy(a, dev_a, a_size, hipMemcpyDeviceToHost);
    hipMemcpy(vec, dev_vec, vec_size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_vec);
    hipFree(dev_c);
}

__device__ float operatorX(int a, int b, float c)
{
    return a * c - b;
}

__global__ void kernel(float* dev_a, float* out_vec, int* c) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    if(j >= N) return;

    __shared__ float row_sum;

    float el = operatorX(i, j, (float) *c);
    dev_a[i * N + j] = el;

    atomicAdd(&row_sum, el);

    if(j == N - 1){
        out_vec[i] = row_sum;
    }
}