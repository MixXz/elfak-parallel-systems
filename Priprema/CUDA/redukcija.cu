#include "hip/hip_runtime.h"
%%cu
#include<iostream>

using namespace std;

#define N 32
#define SIZE 8;

__host__ void init(int *a, int *b, int *c);
__global__ void kernel(int *dev_a, int *dev_b, int *out_c);

int main(void) {
    int a[N], b[N], c[N];

    for(int i = 0; i < N; i++) {
        a[i] = rand() % 5;
        b[i] = rand() % 3;
    }

    init(&a[0], &b[0], &c[0]);

    for(int i = 0 ; i < N; i++) {
        cout << c[i] << " ";
    }
    cout << endl;

    return 0;
}

__host__ void init(int *a, int *b, int *c) {
    int *dev_a, *dev_b, *out_c;
    size_t size = N * sizeof(int);

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&out_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    kernel<<<N / SIZE / 2, SIZE>>>(dev_a, dev_b, out_c);

    hipMemcpy(c, out_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(out_c);
}

__global__ void kernel(int *dev_a, int *dev_b, int *out_c) {
    __shared__ int psum[SIZE];
    int i = threadIdx.x + blockIdx.x * (blockDim.x * 2);

    psum[threadIdx.x] = dev_a[i] * dev_b[i + blockDim.x];
    __syncthreads();

    for(int s = blockDim.x / 2; s > 0; s /= 2) {
        if(threadIdx.x < s) {
            psum[threadIdx.x] += psum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        out_c[blockIdx.x] = psum[0];
    }
}