#include "hip/hip_runtime.h"
%%cu
#include<iostream>

using namespace std;

#define N 32
#define SIZE 8

//Skalarni proizvod dva vektora paralelnom redukcijom.

__host__ void init(int *a, int *b, int *c);
__global__ void kernel(int *dev_a, int *dev_b, int *out_c, const char op);

int main(void) {
    int a[N], b[N], c[N];

    int sum = 0;
    for(int i = 0; i < N; i++) {
        a[i] = rand() % 5;
        b[i] = rand() % 3;
        sum += a[i] * b[i];
    }

    init(&a[0], &b[0], &c[0]);

    cout << c[0] << endl;
    cout << sum << endl;
    return 0;
}

__host__ void init(int *a, int *b, int *c) {
    int *dev_a, *dev_b, *out_c;
    size_t size = N * sizeof(int);

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&out_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    kernel<<<N / SIZE, SIZE>>>(dev_a, dev_b, out_c, '*');
    kernel<<<1, SIZE>>>(out_c, out_c, out_c, '+');

    hipMemcpy(c, out_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(out_c);
}

__global__ void kernel(int *dev_a, int *dev_b, int *out_c, const char op) {
    __shared__ int psum[SIZE];
    int i;

    if(op == '*') {
        i = threadIdx.x + blockIdx.x * blockDim.x;
        psum[threadIdx.x] = dev_a[i] * dev_b[i];
    } else if (op == '+') {
        i = threadIdx.x + blockIdx.x * blockDim.x * 2;
        psum[threadIdx.x] = dev_a[i] + dev_b[i + blockDim.x];
    } else {
        return;
    }
    __syncthreads();

    for(int s = blockDim.x / 2; s > 0; s /= 2) {
        if(threadIdx.x < s) {
            psum[threadIdx.x] += psum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        out_c[blockIdx.x] = psum[0];
    }
}