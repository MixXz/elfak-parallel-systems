#include "hip/hip_runtime.h"
%%cu
#include<iostream>
using namespace std;

#define N 256
#define SIZE 32

__host__ void init(int *a, int *min);
__global__ void extractDiag(int *dev_a, int *out_diag);
__global__ void findMin(int *vec, int *out_min);

int main(void) {
    int a[N][N], min;

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            a[i][j] = rand() % 20;
        }
    }

    init(&a[0][0], &min);

    cout << "Min: " << min << endl;

    return 0;
}

__host__ void init(int *a, int *min) {
    int *dev_a, *out_diag, *out_min;
    size_t size_a = N * N * sizeof(int);
    size_t size_diag = N * sizeof(int);

    hipMalloc(&dev_a, size_a);
    hipMalloc(&out_diag, size_diag);
    hipMalloc(&out_min, sizeof(int));

    hipMemcpy(dev_a, a, size_a, hipMemcpyHostToDevice);

    dim3 grid(ceil(N / SIZE));
    dim3 block(SIZE);
    extractDiag<<<grid, block>>>(dev_a, out_diag);

    dim3 grid2(ceil(N / SIZE / 2));
    findMin<<<grid2, block>>>(out_diag, out_min);

    hipMemcpy(min, out_min, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(out_diag);
    hipFree(out_min);
}

__global__ void extractDiag(int *dev_a, int *out_diag) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= N) {
        return;
    }

    out_diag[tid] = dev_a[tid * N + tid];
}

__global__ void findMin(int *vec, int *out_min) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x * 2; 
 
    if(tid >= N) {
        return;
    }

    __shared__ int partials[SIZE];
    partials[threadIdx.x] = vec[tid + blockDim.x];

    if(vec[tid] < partials[threadIdx.x]) {
        partials[threadIdx.x] = vec[tid];
    }

    for(int stride = blockDim.x / 2; stride > threadIdx.x; stride /= 2) {
        if(partials[threadIdx.x] > partials[threadIdx.x + stride]) {
            partials[threadIdx.x] = partials[threadIdx.x + stride];
        }        
        __syncthreads();
    }

    if(threadIdx.x == 0) {
      *out_min = partials[0];
    }
}