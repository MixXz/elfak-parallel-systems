#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

#define N 8
#define BLOCK_SIZE 32

using namespace std;

__host__ void initProc(int *A, int *B);
__global__ void kernelProc(int *devA, int *outB);

int main(void) {
    int A[N + 2], B[N];

    for(int i = 0; i < N + 2; i++)
        A[i] = 5;
    
    initProc(A, B);

    for(int i = 0; i < N; i++)
        cout << B[i] << " ";

    return 0;
}

__host__ void initProc(int *A, int *B) {
    int* devA, *outB;

    hipMalloc(&devA, (N + 2) * sizeof(int));
    hipMalloc(&outB, N * sizeof(int));

    hipMemcpy(devA, A, (N + 2) * sizeof(int), hipMemcpyHostToDevice);

    kernelProc<<<BLOCK_SIZE, BLOCK_SIZE>>>(devA, outB);

    hipMemcpy(B, outB, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(devA);
    hipFree(outB);
}

__global__ void kernelProc(int *devA, int *outB) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid > N) return;

    __shared__ int shA[BLOCK_SIZE];
    shA[threadIdx.x] = devA[tid];

    __syncthreads();

    if(threadIdx.x < blockDim.x - 2)
        outB[tid] = 0.3 * shA[threadIdx.x] + 0.4 * shA[threadIdx.x + 1] + 0.3 * shA[threadIdx.x + 2];
    else if(threadIdx.x < blockDim.x - 1)
        outB[tid] = 0.3 * shA[threadIdx.x] + 0.4 * shA[threadIdx.x + 1] + 0.3 * devA[tid + 2];
    else
        outB[tid] = 0.3 * shA[threadIdx.x] + 0.4 * devA[tid + 1] + 0.3 * devA[tid + 2];
}