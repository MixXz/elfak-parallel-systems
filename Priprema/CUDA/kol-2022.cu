#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

using namespace std;

#define N 1024
#define P 0.5
#define BLOCK_SIZE 256

__host__ void initProc(int *a, int *b, float *c);
__global__ void kernelProc(int *devA, int *devB, float *outC);

int main(void) {
    int a[N], b[N];
    float c[N - 2];

    for(int i = 0; i < N; i++)
        a[i] = b[i] = i;

    initProc(a, b, c);

    cout << "C: " << endl;
    for(int i = 0; i < N - 2; i++)
        cout << c[i] << " ";

    return 0;
}

__host__ void initProc(int *a, int *b, float *c) {
    int *devA, *devB;
    float *outC;
    size_t sizeAB = N * sizeof(int);
    size_t sizeC = (N - 2) * sizeof(int);

    hipMalloc(&devA, sizeAB);
    hipMalloc(&devB, sizeAB);
    hipMalloc(&outC, sizeC);

    hipMemcpy(devA, a, sizeAB, hipMemcpyHostToDevice);
    hipMemcpy(devB, b, sizeAB, hipMemcpyHostToDevice);

    kernelProc<<<BLOCK_SIZE, BLOCK_SIZE>>>(devA, devB, outC);

    hipMemcpy(c, outC, sizeC, hipMemcpyDeviceToHost);

    hipFree(devA);
    hipFree(devB);
    hipFree(outC);
}

__global__ void kernelProc(int *devA, int *devB, float *outC) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid > N) return;

    __shared__ int shA[BLOCK_SIZE];
    __shared__ int shB[BLOCK_SIZE];

    shA[threadIdx.x] = devA[tid];
    shB[threadIdx.x] = devB[tid];

    __syncthreads();

    if(tid >= N - 2) return;

    if(threadIdx.x < blockDim.x - 2)
        outC[tid] = (shA[threadIdx.x] + shA[threadIdx.x + 1] + shA[threadIdx.x + 2]) * P + (shB[threadIdx.x] + shB[threadIdx.x + 1] + shB[threadIdx.x + 2]) * (1 - P);
    else if (threadIdx.x < blockDim.x - 1)
        outC[tid] = (shA[threadIdx.x] + shA[threadIdx.x + 1] + devA[tid + 2]) * P + (shB[threadIdx.x] + shB[threadIdx.x + 1] + devB[tid + 2]) * (1 - P);
    else
        outC[tid] = (shA[threadIdx.x] + devA[tid + 1] + devA[tid + 2]) * P + (shB[threadIdx.x] + devB[tid + 1] + devB[tid + 2]) * (1 - P);
}